#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <valarray>
#include <sstream>
#include <cstdlib>
#include <hipfft/hipfft.h>
#include <omp.h>
#include <sys/time.h>
#include <queue>
#include "tiffio.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

using namespace std;

typedef complex<double> Cmpx;
int SIGNAL_SIZE = 256 * 256;
float c[256][256];

int dx[4] = { 0, 0, -1, 1 };
int dy[4] = { -1, 1, 0, 0 };

__global__ void border(hipfftComplex* c1, int x, int y)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (((i < 256) && (i >= x) && (j < 256) && (j >= 0)) || ((i < 256) && (i >= 0) && (j < 256) && (j >= y))) {
        c1[i * 256 + j].x = 0;
    }
}

__global__ void dilate(hipfftComplex* c1, hipfftComplex* c2)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < 255) && (j < 255) && (i >= 1) && (j >= 1)) {
        if (c1[i * 256 + j].x > 200) {
            c2[i * 256 + j].x = c1[i * 256 + j].x;
            c2[i * 256 + j].x = 255;
            c2[(i - 1) * 256 + j].x = 255;
            c2[(i - 1) * 256 + j + 1].x = 255;
            c2[i * 256 + j - 1].x = 255;
            c2[i * 256 + j + 1].x = 255;
            c2[(i - 1) * 256 + j - 1].x = 255;
            c2[(i + 1) * 256 + j + 1].x = 255;
            c2[(i - 1) * 256 + j + 1].x = 255;
            c2[(i + 1) * 256 + j - 1].x = 255;
        }
    }
}

__global__ void sobel(hipfftComplex* c1, hipfftComplex* c2)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < 254) && (j < 254) && (i >= 0) && (j >= 0)) {

        float Gx = (2 * c1[(i + 2) * 256 + j + 1].x + c1[(i + 2) * 256 + j].x + c1[(i + 2) * 256 + j + 2].x) - (2 * c1[i * 256 + j + 1].x + c1[i * 256 + j].x + c1[i * 256 + j + 2].x);
        float Gy = (2 * c1[(i + 1) * 256 + j + 2].x + c1[i * 256 + j + 2].x + c1[(i + 2) * 256 + j + 2].x) - (2 * c1[(i + 1) * 256 + j].x + c1[i * 256 + j].x + c1[(i + 2) * 256 + j].x);
        c2[i * 256 + j].x = sqrt(Gx * Gx + Gy * Gy);
        if (c2[i * 256 + j].x < 70)
            c2[i * 256 + j].x = 0;
        else
            c2[i * 256 + j].x = 255;
    }
}

__global__ void divide(hipfftComplex* c1, int SIGNAL_SIZE)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i < 256) && (j < 256)) {

        c1[i * 256 + j].x = c1[i * 256 + j].x / (float)SIGNAL_SIZE;
        c1[i * 256 + j].y = c1[i * 256 + j].y / (float)SIGNAL_SIZE;
    }
}

__global__ void filter(hipfftComplex* d_signal)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((j < 256) && (j >= 3)) {
        if (((i < 10) && (i >= 0)) || ((i < 256) && (i >= 250))) {
            d_signal[i * 256 + j].x = 0;
            d_signal[j * 256 + i].x = 0;
            d_signal[i * 256 + j].y = 0;
            d_signal[j * 256 + i].y = 0;
        }
    }
}

int main()
{
    TIFF* tif = TIFFOpen("aaaa.tif", "r");

    struct timeval t1, t2;
    gettimeofday(&t1, NULL);
    int comp = 0;

    int mem_size = sizeof(hipfftComplex) * SIGNAL_SIZE;
    // Allocate device memory for signal
    hipfftComplex* d_signal;
    hipfftComplex* d_signal1;
    hipfftComplex* d_signal2;
    checkCudaErrors(hipMalloc((void**)&d_signal, mem_size));
    checkCudaErrors(hipMalloc((void**)&d_signal1, mem_size));
    checkCudaErrors(hipMalloc((void**)&d_signal2, mem_size));
    hipfftHandle plan, invplan;
    hipfftPlan2d(&plan, 256, 256, HIPFFT_C2C);
    hipfftPlan2d(&invplan, 256, 256, HIPFFT_C2C);
    hipfftComplex* h_inverse_signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SIGNAL_SIZE);
    hipfftComplex* h_inverse_signal2 = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SIGNAL_SIZE);

    do {
        uint32 imagelength;
        tsize_t scanline;
        tdata_t buf;
        uint32 row;
        uint32 col;
        uint8* data;

        TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &imagelength);
        scanline = TIFFScanlineSize(tif);
        buf = _TIFFmalloc(scanline);
        for (row = 0; row < imagelength; row++) {
            TIFFReadScanline(tif, buf, row);
            for (col = 0; col < scanline; col++) {
                data = (uint8*)buf;

                c[row][col] = data[col];
            }
        }
        // #pragma omp parallel for schedule(dynamic) collapse(2)
        for (int i = 0; i < 256; i++)
            for (int j = 0; j < 256; j++)
                if ((i >= imagelength) || (j >= scanline))
                    c[i][j] = 0;

        hipfftComplex* h_signal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SIGNAL_SIZE);

        // #pragma omp parallel for schedule(dynamic) collapse(2)
        for (unsigned int k1 = 0; k1 < 256; ++k1) {
            for (int k2 = 0; k2 < 256; k2++) {
                h_signal[k1 * 256 + k2].x = c[k1][k2];
                h_signal[k1 * 256 + k2].y = 0;
            }
        }

        // Copy host memory to device
        checkCudaErrors(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));

        // CUFFT plan

        // Transform signal
        // printf("Transforming signal hipfftExecC2C\n");
        checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex*)d_signal, (hipfftComplex*)d_signal, HIPFFT_FORWARD));

        dim3 nbthreadsbyblock(32, 32);
        dim3 nbblocksbygrid(128, 128);
        filter<<<nbblocksbygrid, nbthreadsbyblock> > >(d_signal);

        // Transform signal back
        // printf("Transforming signal back hipfftExecC2C\n");
        checkCudaErrors(hipfftExecC2C(invplan, (hipfftComplex*)d_signal, (hipfftComplex*)d_signal, HIPFFT_BACKWARD));

        divide<<<nbblocksbygrid, nbthreadsbyblock> > >(d_signal, SIGNAL_SIZE);
        sobel<<<nbblocksbygrid, nbthreadsbyblock> > >(d_signal, d_signal1);
        dilate<<<nbblocksbygrid, nbthreadsbyblock> > >(d_signal1, d_signal2);
        border<<<nbblocksbygrid, nbthreadsbyblock> > >(d_signal2, imagelength, scanline);

        // Copy device memory to host

        checkCudaErrors(hipMemcpy(h_inverse_signal, d_signal2, mem_size, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_inverse_signal2, d_signal2, mem_size, hipMemcpyDeviceToHost));

        queue<pair<int, int> > Q;
        Q.push(make_pair(250, 250));
        while (!Q.empty()) {
            pair<int, int> p = Q.front();
            Q.pop();
            for (int i = 0; i < 4; i++) {
                if ((p.first + dx[i] >= 0) && (p.first + dx[i] <= 255) && (p.second + dy[i] >= 0) && (p.second + dy[i] <= 255)) {
                    if (h_inverse_signal2[(p.first + dx[i]) * 256 + p.second + dy[i]].x < 255) {
                        h_inverse_signal2[(p.first + dx[i]) * 256 + p.second + dy[i]].x = 255;
                        Q.push(make_pair(p.first + dx[i], p.second + dy[i]));
                    }
                }
            }
        }
        //#pragma omp parallel for schedule(dynamic) collapse(2)
        for (int i = 0; i < 255; i++) {
            for (int j = 0; j < 255; j++) {
                if ((unsigned int)h_inverse_signal2[i * 256 + j].x < 255) {
                    h_inverse_signal[i * 256 + j].x = 255;
                }
                if (h_inverse_signal[i * 256 + j].x != 255) {
                    c[i][j] = 180;
                }
            }
        }

        char* m_data_cropped = (char*)malloc(sizeof(char) * scanline * imagelength * 3);

        //#pragma omp parallel for schedule(dynamic) collapse(2)
        for (int i = 0; i < imagelength; i++) {
            for (int j = 0; j < scanline; j++) {
                if (h_inverse_signal2[i * 256 + j].x == 0) {
                    m_data_cropped[3 * (i * scanline + j)] = 100;
                    m_data_cropped[3 * (i * scanline + j) + 1] = 0;
                    m_data_cropped[3 * (i * scanline + j) + 2] = 0;
                }
                else {
                    m_data_cropped[3 * (i * scanline + j)] = c[i][j];
                    m_data_cropped[3 * (i * scanline + j) + 1] = c[i][j];
                    m_data_cropped[3 * (i * scanline + j) + 2] = c[i][j];
                }
            }
        }

        string s1 = to_string(comp);
        char const* pchar = s1.c_str();
        char s[20];
        strcpy(s, pchar);
        strcat(s, ".tif");

        TIFF* tif1 = TIFFOpen(s, "w");
        TIFFSetField(tif1, TIFFTAG_IMAGEWIDTH, scanline);
        TIFFSetField(tif1, TIFFTAG_IMAGELENGTH, imagelength);
        TIFFSetField(tif1, TIFFTAG_SAMPLESPERPIXEL, 3);
        TIFFSetField(tif1, TIFFTAG_BITSPERSAMPLE, 8);
        TIFFSetField(tif1, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);

        if ((TIFFWriteEncodedStrip(tif1, 0, &m_data_cropped[0], scanline * imagelength * 3)) == -1) {
            //std::cerr << "Unable to write tif file: " << "image.tif" << std::endl;
        }
        else {
            //std::cout << "Image is saved! size is : " << image_s << std::endl;
        }

        TIFFClose(tif1);
        comp++;
        TIFFWriteDirectory(tif);
    } while (comp < 2000);

    hipFree(d_signal);
    hipFree(d_signal1);
    hipFree(d_signal2);
    gettimeofday(&t2, NULL);

    double duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
    cout << "duration= " << duration << endl;

    return 0;
}

